#include "hip/hip_runtime.h"
#include "cuda_functions.cuh"
#include "compute_functions.cuh"
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>

#define CUDA_RT_CALL(call) { \
hipError_t cudaStatus = call; \
        if (hipSuccess != cudaStatus) {\
            fprintf(stderr,\
            "ERROR: CUDA RT call \"%s\" in line %d of file %s failed"\
                    "with "\
                    "%s (%d).\n",\
                    #call, __LINE__, __FILE__,hipGetErrorString(cudaStatus), cudaStatus); \
            throw std::runtime_error("CUDA ERROR"); \
        }\
    }


inline int32_t cudaMemChk(hipError_t error) {
	if (error != hipSuccess) {
		fprintf(stderr, "Erreur allocation CUDA\n");
		return EXIT_FAILURE;
	} else
		return EXIT_SUCCESS;
}

bool run_configCUDA(const char *filename,
					float **matrix,
					size_t *matCol,
					size_t *matRow,
					heatPoint **srcsHeat,
					size_t *srcsSize,
					uint32_t *numIter,
					uint32_t *sortieImage) {
	//Vérif partielle.
	int64_t numHeatPnt = 0;
	FILE *file = NULL;
	file = fopen(filename, "r");

	if (file == NULL) {
		perror("Erreur fopen : ");
		return false;
	}

	if (fscanf(file, "%zu", matCol) == EOF ||
		fscanf(file, "%zu", matRow) == EOF ||
		fscanf(file, "%u", numIter) == EOF ||
		fscanf(file, "%u", sortieImage) == EOF ||
		fscanf(file, "%li", &numHeatPnt) == EOF) {
		perror("Erreur fscanf : ");
		fclose(file);
		return false;
	}

	if (numHeatPnt <= 0) {
		fclose(file);
		return false;
	}

	*srcsSize = numHeatPnt;

	if (*matCol <= 0) *matCol = 1000;
	if (*matRow <= 0) *matRow = 1000;

	if ((*matrix = (float *) calloc(*matRow * *matCol, sizeof(float))) == NULL) {
		perror("Erreir calloc matrices : ");
		fclose(file);
		return false;
	}

	if ((*srcsHeat = (heatPoint *) calloc(*srcsSize, sizeof(heatPoint))) == NULL) {
		perror("Error malloc heatPoints :");
		free(*matrix);
		fclose(file);
		return false;
	}

	if (*sortieImage > *numIter)
		*sortieImage %= *numIter;

	for (int64_t i = 0; i < numHeatPnt; ++i) {
		int64_t m = 0, n = 0;
		size_t x, y;
		if (fscanf(file, "%li", &m) == EOF || fscanf(file, "%li", &n) == EOF) {
			fclose(file);
			free(matrix);
			return false;
		}
		if (m < 0 || m >= *matRow || n < 0 || n >= *matCol) {
			fclose(file);
			free(matrix);
			return false;
		}
		(*srcsHeat)[i].x = x = (size_t) n, (*srcsHeat)[i].y = y = (size_t) m;
		/*
		 * Les coordonnées données dans le fichier de configuration servent à décrire le milieu du point de chaleur (c'est un carré)
		 */
		(*matrix)[offset(x, y, *matRow)] = 1.0f;

		(*matrix)[offset(x, y + 1, *matRow)] = 1.0f;

		(*matrix)[offset(x + 1, y, *matRow)] = 1.0f;

		(*matrix)[offset(x + 1, y + 1, *matRow)] = 1.0f;

		(*matrix)[offset(x - 1, y, *matRow)] = 1.0f;

		(*matrix)[offset(x, y - 1, *matRow)] = 1.0f;

		(*matrix)[offset(x - 1, y - 1, *matRow)] = 1.0f;

		(*matrix)[offset(x - 1, y + 1, *matRow)] = 1.0f;

		(*matrix)[offset(x + 1, y - 1, *matRow)] = 1.0f;

	}

	return fclose(file) == 0;
}

int32_t run_cuda(float *h_matrix,
				 size_t matCol,
				 size_t matRow,
				 heatPoint *h_srcs,
				 size_t srcSize,
				 uint32_t numIter,
				 uint32_t sortieImage,
				 CvMat *img,
				 float convergence) {
	uint32_t numThread = 16;
	float *d_val = NULL, *d_val_new = NULL, kernelTime = 0.f;
	heatPoint *d_srcs = NULL;
	hipEvent_t start, stop;

	hipEventCreate(&start), hipEventCreate(&stop);

	if (cudaMemChk(hipMalloc((void **) &d_val, matCol * matRow * sizeof(float))) == EXIT_FAILURE) {
		end_simulation(h_matrix, h_srcs);
		return EXIT_FAILURE;
	}

	if (cudaMemChk(hipMalloc((void **) &d_val_new, matCol * matRow * sizeof(float))) == EXIT_FAILURE) {
		hipFree(d_val);
		end_simulation(h_matrix, h_srcs);
		return EXIT_FAILURE;
	}

	if (cudaMemChk(hipMalloc((void **) &d_srcs, srcSize * sizeof(heatPoint))) == EXIT_FAILURE) {
		hipFree(d_val);
		hipFree(d_val_new);
		end_simulation(h_matrix, h_srcs);
		return EXIT_FAILURE;
	}

	if (cudaMemChk(hipMemcpy(d_val, h_matrix, matCol * matRow * sizeof(float), hipMemcpyHostToDevice)) == EXIT_FAILURE) {
		hipFree(d_val);
		hipFree(d_val_new);
		hipFree(d_srcs);
		end_simulation(h_matrix, h_srcs);
		return EXIT_FAILURE;
	}

	if (cudaMemChk(hipMemcpy(d_srcs, h_srcs, srcSize * sizeof(heatPoint), hipMemcpyHostToDevice)) == EXIT_FAILURE) {
		hipFree(d_val);
		hipFree(d_val_new);
		hipFree(d_srcs);
		end_simulation(h_matrix, h_srcs);
		return EXIT_FAILURE;
	}

	if (cudaMemChk(hipMemset(d_val_new, 0, matCol * matRow * sizeof(float))) == EXIT_FAILURE) {
		hipFree(d_val);
		hipFree(d_val_new);
		hipFree(d_srcs);
		end_simulation(h_matrix, h_srcs);
		return EXIT_FAILURE;
	}

	//À optimiser avec les defines en fonction du GPU (en dur pour le Kepler pour l'instant)
	dim3 dimGrid;
	dimGrid.x = (uint32_t) ceil((matCol - 1.0) / numThread);
	dimGrid.y = (uint32_t) ceil((matRow - 1.0) / numThread);
	dimGrid.z = 1;

	dim3 dimBlock;
	dimBlock.x = numThread;
	dimBlock.y = numThread;
	dimBlock.z = 1;

	//Lancement du chrono
	hipEventRecord(start);
	simulationKernel << < dimGrid, dimBlock >> > (d_val_new, d_val, matCol, matRow, convergence, numIter, d_srcs, srcSize);
	hipEventRecord(stop); //Arrêt

	if (cudaMemChk(hipMemcpy(h_matrix, d_val, matCol * matRow * sizeof(float), hipMemcpyDeviceToHost)) == EXIT_FAILURE) {
		fprintf(stderr, "Transfert du résultat impossible.\n");
		hipFree(d_val);
		hipFree(d_val_new);
		hipFree(d_srcs);
		end_simulation(h_matrix, h_srcs);
		return EXIT_SUCCESS;
	}
	hipEventSynchronize(stop);

	hipEventElapsedTime(&kernelTime, start, stop);

	printf("Temps de la simulation : %f\n", kernelTime);

	hipFree(d_val);
	hipFree(d_val_new);
	hipFree(d_srcs);
	end_simulation(h_matrix, h_srcs);
	return EXIT_SUCCESS;
}

void end_simulation(float *__restrict h_matrix, heatPoint *__restrict h_srcs) {
	free(h_matrix);
	free(h_srcs);
	puts("Libération mémoire");
}